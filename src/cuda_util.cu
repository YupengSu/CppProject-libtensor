#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstddef>
#include <cstdio>
#include <iostream>
#include <vector>

#include "config.hpp"
#include "cuda_util.cuh"
#include "data_type.cuh"
#include "size.hpp"
#include "serial_tensor.hpp"

namespace ts {
void checkCudaError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << file
                  << ":" << line << std::endl;
        exit(1);
    }
}

#define checkCudaError(err) checkCudaError(err, __FILE__, __LINE__)

__device__ void add_data_t(data_t& dst, data_t& a, data_t& b) {
    switch (a.dtype) {
        case dt::int8:
            dst.data.tensor_int8 = a.data.tensor_int8 + b.data.tensor_int8;
        break;
        case dt::float32:
            dst.data.tensor_float32 =
                a.data.tensor_float32 + b.data.tensor_float32;
        break;
        case dt::bool8:
            dst.data.tensor_bool = a.data.tensor_bool + b.data.tensor_bool;
        break;
        case dt::int32:
            dst.data.tensor_int32 = a.data.tensor_int32 + b.data.tensor_int32;
        break;
        case dt::float64:
            dst.data.tensor_float64 =
                a.data.tensor_float64 + b.data.tensor_float64;
        break;
    }
    dst.dtype=a.dtype;
}
__global__ void addMMKernel(data_t* c, data_t* a, data_t* b, size_t size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        add_data_t(c[i], a[i], b[i]);
    }

}

__device__ void get_idx(size_t &dst, size_t index, int * shape, int * stride, int dim) {
    dst = 0;
    for (int i = 0; i < dim; i++) {
        dst += index / stride[i] % shape[i] * stride[i];
    }
}

__global__ void addTensorKernel(data_t* c, data_t* a, data_t* b, size_t size, int * shape, int * stride, int dim) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        size_t offset;
        get_idx(offset, i, shape, stride, dim);
        // offset = i;
        add_data_t(c[i], a[offset], b[offset]);
    }
}
__global__ void addTensorKernelNum(data_t* c, data_t* a, data_t b, size_t size, int * shape, int * stride, int dim) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        size_t offset;
        get_idx(offset, i, shape, stride, dim);
        // offset = i;
        add_data_t(c[i], a[offset], b);
    }
}


extern void c_cudaMalloc(void** ptr, size_t size) {
    checkCudaError(hipMalloc(ptr, size));
}

extern void c_cudaMemcpy(void* dst, void* src, size_t size,
                         c_cudaMemcpyKind kind) {
    checkCudaError(hipMemcpy(dst, src, size, (hipMemcpyKind)kind));
}

extern void c_cudaFree(void* src) { checkCudaError(hipFree(src)); }

extern void addMM(void* c, void* a, void* b, size_t size) {
    data_t* dev_c = (data_t*)c;
    data_t* dev_a = (data_t*)a;
    data_t* dev_b = (data_t*)b;
    size_t bytes = size * sizeof(data_t);
    size_t threadsPerBlock = THREAD_PER_BLOCK;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addMMKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, dev_b, size);

    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}


extern void addKernel(void* dst, Tensor a, Tensor b, size_t size) {
    data_t* dev_a = (data_t*)a.data.dp;
    data_t* dev_b = (data_t*)b.data.dp;
    data_t* dev_c = (data_t*)dst;
    size_t bytes = size * sizeof(data_t);
    size_t threadsPerBlock = THREAD_PER_BLOCK;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int * shape;
    int * stide;
    checkCudaError(hipMalloc(&shape, a.shape.shape.size() * sizeof(int)));
    checkCudaError(hipMalloc(&stide, a.stride.size() * sizeof(int)));
    checkCudaError(hipMemcpy(shape, a.shape.shape.data(), a.shape.shape.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(stide, a.stride.data(), a.stride.size() * sizeof(int), hipMemcpyHostToDevice));

    addTensorKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, dev_b,  size, shape, stide, a.get_dim());

    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}

extern void addKernelNum(void *dst, Tensor a, data_t b, size_t size) {
    data_t* dev_a = (data_t*)a.data.dp;
    data_t* dev_c = (data_t*)dst;
    b = b.to_dt(a.dtype);
    size_t bytes = size * sizeof(data_t);
    size_t threadsPerBlock = THREAD_PER_BLOCK;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int * shape;
    int * stide;
    checkCudaError(hipMalloc(&shape, a.shape.shape.size() * sizeof(int)));
    checkCudaError(hipMalloc(&stide, a.stride.size() * sizeof(int)));
    checkCudaError(hipMemcpy(shape, a.shape.shape.data(), a.shape.shape.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(stide, a.stride.data(), a.stride.size() * sizeof(int), hipMemcpyHostToDevice));

    addTensorKernelNum<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, b,  size, shape, stide, a.get_dim());

    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}


}  // namespace ts
