#include "hip/hip_runtime.h"
#include "cuda_util.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

#define THREAD_PER_BLOCK 256

namespace ts {
    void checkCudaError(hipError_t err, const char* file, int line) {
        if (err != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << file << ":" << line << std::endl;
            exit(1);
        }
    }
    
    #define checkCudaError(err) checkCudaError(err, __FILE__, __LINE__)
    
    __global__ void addMMKernel(double* c, double* a, double* b, int size) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            c[i] = a[i] + b[i];
        }
    }
    
    extern void addMM(void* c, void* a, void* b, int size) {
        double* dev_c;
        double* dev_a;
        double* dev_b;
        size_t bytes = size * sizeof(double);
    
        checkCudaError(hipMalloc(&dev_c, bytes));
        checkCudaError(hipMalloc(&dev_a, bytes));
        checkCudaError(hipMalloc(&dev_b, bytes));
        
        checkCudaError(hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice));
        checkCudaError(hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice));

        size_t threadsPerBlock = THREAD_PER_BLOCK;
        size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        addMMKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_c, dev_a, dev_b, size);

        checkCudaError(hipGetLastError());
        checkCudaError(hipDeviceSynchronize());

        checkCudaError(hipMemcpy(c, dev_c, bytes, hipMemcpyDeviceToHost));

        checkCudaError(hipFree(dev_c));
        checkCudaError(hipFree(dev_a));
        checkCudaError(hipFree(dev_b));
    }
}

